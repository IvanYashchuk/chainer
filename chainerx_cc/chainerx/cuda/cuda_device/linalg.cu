#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/native/native_device.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/linalg.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
hipsolverStatus_t GeqrfBufferSize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by geqrf (QR)"};
}

template <typename T>
hipsolverStatus_t Geqrf(
        hipsolverHandle_t /*handle*/,
        int /*m*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*tau*/,
        T* /*workspace*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by geqrf (QR)"};
}

template <typename T>
hipsolverStatus_t OrgqrBufferSize(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, int /*k*/, T* /*a*/, int /*lda*/, T* /*tau*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by orgqr (QR)"};
}

template <typename T>
hipsolverStatus_t Orgqr(
        hipsolverHandle_t /*handle*/,
        int /*m*/,
        int /*n*/,
        int /*k*/,
        T* /*a*/,
        int /*lda*/,
        T* /*tau*/,
        T* /*work*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by orgqr (QR)"};
}

template <>
hipsolverStatus_t GeqrfBufferSize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgeqrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GeqrfBufferSize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgeqrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Geqrf<double>(
        hipsolverHandle_t handle, int m, int n, double* a, int lda, double* tau, double* workspace, int lwork, int* devinfo) {
    return hipsolverDnDgeqrf(handle, m, n, a, lda, tau, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t Geqrf<float>(
        hipsolverHandle_t handle, int m, int n, float* a, int lda, float* tau, float* workspace, int lwork, int* devinfo) {
    return hipsolverDnSgeqrf(handle, m, n, a, lda, tau, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t OrgqrBufferSize<double>(hipsolverHandle_t handle, int m, int n, int k, double* a, int lda, double* tau, int* lwork) {
    return hipsolverDnDorgqr_bufferSize(handle, m, n, k, a, lda, tau, lwork);
}

template <>
hipsolverStatus_t OrgqrBufferSize<float>(hipsolverHandle_t handle, int m, int n, int k, float* a, int lda, float* tau, int* lwork) {
    return hipsolverDnSorgqr_bufferSize(handle, m, n, k, a, lda, tau, lwork);
}

template <>
hipsolverStatus_t Orgqr<double>(
        hipsolverHandle_t handle, int m, int n, int k, double* a, int lda, double* tau, double* work, int lwork, int* devinfo) {
    return hipsolverDnDorgqr(handle, m, n, k, a, lda, tau, work, lwork, devinfo);
}

template <>
hipsolverStatus_t Orgqr<float>(
        hipsolverHandle_t handle, int m, int n, int k, float* a, int lda, float* tau, float* work, int lwork, int* devinfo) {
    return hipsolverDnSorgqr(handle, m, n, k, a, lda, tau, work, lwork, devinfo);
}

template <typename T>
std::tuple<Array, Array> QRImpl(const Array& a, QRMode mode) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    int64_t m = a.shape()[0];
    int64_t n = a.shape()[1];
    int64_t mn = std::min(m, n);

    Array Q = Empty(Shape{0}, dtype, device);
    Array R = a.Transpose().Copy();  // QR decomposition is done in-place
    Array tau = Empty(Shape{mn}, dtype, device);

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    T* r_ptr = static_cast<T*>(internal::GetRawOffsetData(R));
    T* tau_ptr = static_cast<T*>(internal::GetRawOffsetData(tau));

    std::shared_ptr<void> devInfo = device.Allocate(sizeof(int));

    int buffersize_geqrf = 0;
    device_internals.cusolverdn_handle().Call(GeqrfBufferSize<T>, m, n, r_ptr, n, &buffersize_geqrf);

    Array work = Empty(Shape{buffersize_geqrf}, dtype, device);
    T* work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    device_internals.cusolverdn_handle().Call(
            Geqrf<T>, m, n, r_ptr, m, tau_ptr, work_ptr, buffersize_geqrf, static_cast<int*>(devInfo.get()));

    int devInfo_h = 0;
    Device& native_device = dynamic_cast<native::NativeDevice&>(GetDefaultContext().GetDevice({"native", 0}));
    device.MemoryCopyTo(&devInfo_h, devInfo.get(), sizeof(int), native_device);
    if (devInfo_h != 0) {
        throw ChainerxError{"Unsuccessfull geqrf (QR) execution. Info = ", devInfo_h};
    }

    if (mode == QRMode::r) {
        R = R.At(std::vector<ArrayIndex>{Slice{}, Slice{0, mn}}).Transpose();  // R = R[:, 0:mn].T
        R = Triu(R, 0);
        return std::make_tuple(std::move(Q), std::move(R));
    }

    if (mode == QRMode::raw) {
        return std::make_tuple(std::move(R), std::move(tau));
    }

    int64_t mc;
    if (mode == QRMode::complete && m > n) {
        mc = m;
        Q = Empty(Shape{m, m}, dtype, device);
    } else {
        mc = mn;
        Q = Empty(Shape{n, m}, dtype, device);
    }

    device.backend().CallKernel<CopyKernel>(R, Q.At(std::vector<ArrayIndex>{Slice{0, n}, Slice{}}));  // Q[0:n, :] = R
    T* q_ptr = static_cast<T*>(internal::GetRawOffsetData(Q));

    int buffersize_orgqr = 0;
    device_internals.cusolverdn_handle().Call(OrgqrBufferSize<T>, m, mc, mn, q_ptr, m, tau_ptr, &buffersize_orgqr);

    work = Empty(Shape{buffersize_orgqr}, dtype, device);

    device_internals.cusolverdn_handle().Call(
            Orgqr<T>, m, mc, mn, q_ptr, m, tau_ptr, work_ptr, buffersize_orgqr, static_cast<int*>(devInfo.get()));

    device.MemoryCopyTo(&devInfo_h, devInfo.get(), sizeof(int), native_device);
    if (devInfo_h != 0) {
        throw ChainerxError{"Unsuccessfull orgqr (QR) execution. Info = ", devInfo_h};
    }

    // .Copy() is needed to have correct strides
    Q = Q.At(std::vector<ArrayIndex>{Slice{0, mc}, Slice{}}).Transpose().Copy();  // Q = Q[0:mc, :].T
    R = R.At(std::vector<ArrayIndex>{Slice{}, Slice{0, mc}}).Transpose();  // R = R[:, 0:mc].T
    R = Triu(R, 0);
    return std::make_tuple(std::move(Q), std::move(R));
}

template <typename T>
hipsolverStatus_t GetrfBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrf(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, T* /*workspace*/, int* /*devipiv*/, int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrs(
        hipsolverHandle_t /*handle*/,
        hipblasOperation_t /*trans*/,
        int /*n*/,
        int /*nrhs*/,
        T* /*a*/,
        int /*lda*/,
        int* /*devipiv*/,
        T* /*b*/,
        int /*ldb*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrs (Solve)"};
}

template <>
hipsolverStatus_t GetrfBuffersize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GetrfBuffersize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Getrf<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, double* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnDgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrf<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, float* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnSgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrs<double>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        double* a,
        int lda,
        int* devipiv,
        double* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnDgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t Getrs<float>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        float* a,
        int lda,
        int* devipiv,
        float* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnSgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <typename T>
void SolveImpl(const Array& a, const Array& b, const Array& out) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    Array lu_matrix = Empty(a.shape(), dtype, device);
    device.backend().CallKernel<CopyKernel>(a.Transpose(), lu_matrix);
    T* lu_ptr = static_cast<T*>(internal::GetRawOffsetData(lu_matrix));

    int m = a.shape()[0];
    int nrhs = 1;
    if (b.ndim() == 2) {
        nrhs = b.shape()[1];
    }

    Array ipiv = Empty(Shape{m}, Dtype::kInt32, device);
    int* ipiv_ptr = static_cast<int*>(internal::GetRawOffsetData(ipiv));

    int buffersize = 0;
    device_internals.cusolverdn_handle().Call(GetrfBuffersize<T>, m, m, lu_ptr, m, &buffersize);

    Array work = Empty(Shape{buffersize}, dtype, device);
    T* work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

    device_internals.cusolverdn_handle().Call(Getrf<T>, m, m, lu_ptr, m, work_ptr, ipiv_ptr, static_cast<int*>(devinfo.get()));

    int devinfo_h = 0;
    Device& native_device = dynamic_cast<native::NativeDevice&>(GetDefaultContext().GetDevice({"native", 0}));
    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessfull getrf (LU) execution. Info = ", devinfo_h};
    }

    Array out_transposed = b.Transpose().Copy();
    T* out_ptr = static_cast<T*>(internal::GetRawOffsetData(out_transposed));

    device_internals.cusolverdn_handle().Call(
            Getrs<T>, HIPBLAS_OP_N, m, nrhs, lu_ptr, m, ipiv_ptr, out_ptr, m, static_cast<int*>(devinfo.get()));

    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessfull getrs (Solve) execution. Info = ", devinfo_h};
    }

    device.backend().CallKernel<CopyKernel>(out_transposed.Transpose(), out);
}

}  // namespace

class CudaQRKernel : public QRKernel {
public:
    std::tuple<Array, Array> Call(const Array& a, QRMode mode = QRMode::reduced) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        return VisitFloatingPointDtype(dtype, [&](auto pt) -> std::tuple<Array, Array> {
            using T = typename decltype(pt)::type;
            return QRImpl<T>(a, mode);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(QRKernel, CudaQRKernel);

class CudaSolveKernel : public SolveKernel {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        VisitFloatingPointDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            SolveImpl<T>(a, b, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SolveKernel, CudaSolveKernel);

class CudaInverseKernel : public InverseKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        // There is LAPACK routine ``getri`` for computing the inverse of an LU-factored matrix,
        // but cuSOLVER does not have it implemented, therefore inverse is obtained with ``getrs``
        // inv(A) == solve(A, Identity)
        Array b = Identity(a.shape()[0], dtype, device);
        device.backend().CallKernel<SolveKernel>(a, b, out);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(InverseKernel, CudaInverseKernel);

}  // namespace cuda
}  // namespace chainerx
